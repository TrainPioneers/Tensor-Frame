
#include <hip/hip_runtime.h>
// Matrix multiplication kernels for CUDA
extern "C" {

// Basic matrix multiplication kernel
// C = A * B where A is MxK, B is KxN, C is MxN
__global__ void matmul_kernel(const float* A, const float* B, float* C, 
                              int M, int N, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < M && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < K; k++) {
            sum += A[row * K + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

// Optimized matrix multiplication kernel using shared memory
__global__ void matmul_shared_kernel(const float* A, const float* B, float* C, 
                                     int M, int N, int K) {
    const int TILE_SIZE = 16;
    __shared__ float As[16][16];
    __shared__ float Bs[16][16];
    
    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;
    
    int row = by * TILE_SIZE + ty;
    int col = bx * TILE_SIZE + tx;
    
    float sum = 0.0f;
    
    for (int tile = 0; tile < (K + TILE_SIZE - 1) / TILE_SIZE; ++tile) {
        // Load tiles into shared memory
        if (row < M && tile * TILE_SIZE + tx < K) {
            As[ty][tx] = A[row * K + tile * TILE_SIZE + tx];
        } else {
            As[ty][tx] = 0.0f;
        }
        
        if (col < N && tile * TILE_SIZE + ty < K) {
            Bs[ty][tx] = B[(tile * TILE_SIZE + ty) * N + col];
        } else {
            Bs[ty][tx] = 0.0f;
        }
        
        __syncthreads();
        
        // Compute partial dot product
        for (int k = 0; k < TILE_SIZE; ++k) {
            sum += As[ty][k] * Bs[k][tx];
        }
        
        __syncthreads();
    }
    
    if (row < M && col < N) {
        C[row * N + col] = sum;
    }
}

} // extern "C"